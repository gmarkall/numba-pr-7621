#include <iostream>
#include "caller.cuh"


int main(int __attribute__((unused)) argc, char __attribute__((unused)) *argv[])
{
  float *m_r, *m_x, *m_y;
  hipMallocManaged(&m_x, sizeof(float));
  hipMallocManaged(&m_y, sizeof(float));
  hipMallocManaged(&m_r, sizeof(float));

  *m_x = 3.0f;
  *m_y = 2.0f;
  
  hipDeviceSynchronize();
  caller<<<1, 1>>>(m_r, m_x, m_y);
  hipDeviceSynchronize();

  // Expectation: (3 * 2) + 1 = 7
  std::cout << "Result is " << *m_r << std::endl;

  hipFree(m_r);
  hipFree(m_x);
  hipFree(m_y);
}
