#include "hip/hip_runtime.h"
#include "functions.cuh"
#include "caller.cuh"

__global__ void caller(float *r, float *x, float *y)
{
  mul_f32_f32(r, *x, *y);
}
