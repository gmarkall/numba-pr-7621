#include "hip/hip_runtime.h"
#include "functions.cuh"
#include "caller.cuh"

__global__ void caller(float *r, float *x, float *y)
{
  float r_l[1];
  mul_f32_f32(r_l, *x, *y);
  r[0] = r_l[0];
}
