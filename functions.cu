#include "hip/hip_runtime.h"
#include "functions.cuh"

extern "C" __device__ int
mul_f32_f32(
  float* return_value,
  float x,
  float y)
{
  // Compute result and store in caller-provided slot
  *return_value = x * y;
  atomicAdd(return_value, 1.0f);

  // Signal that no Python exception occurred
  return 0;
}

